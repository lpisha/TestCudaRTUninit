// ======================================================================== //
// Copyright 2022 Louis Pisha                                               //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cinttypes>
#include <thread>

#define checkCudaErrors(val) checkCudaErrors_( (val), #val, __LINE__ )
void checkCudaErrors_(hipError_t result, char const *const func, int const line)
{
    if(result){
        std::cout << "CUDA call " << func << " failed line " << line << "\n";
        hipDeviceReset();
        std::abort();
    }
}

const char * const memTypeNames[4] = {
    "hipMemoryTypeUnregistered",
    "hipMemoryTypeHost",
    "hipMemoryTypeDevice",
    "hipMemoryTypeManaged"
};
const char *getMemType(int i){
    if(i >= 0 && i < 4) return memTypeNames[i];
    return "error";
}

float *mem_d;

void threadfunc()
{
    int dev;
    checkCudaErrors(hipGetDevice(&dev));
    printf("CUDA device is %d\n", dev);
    hipPointerAttribute_t attrs;
    checkCudaErrors(hipPointerGetAttributes(&attrs, mem_d));
    printf("type %s, device %d, devptr %016" PRIx64 ", hostptr %016" PRIx64 "\n",
        getMemType((int)attrs.type), attrs.device, attrs.devicePointer, attrs.hostPointer);
    printf("Setting CUDA device to 0\n");
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipPointerGetAttributes(&attrs, mem_d));
    printf("type %s, device %d, devptr %016" PRIx64 ", hostptr %016" PRIx64 "\n",
        getMemType((int)attrs.type), attrs.device, attrs.devicePointer, attrs.hostPointer);
}


int main(int argc, char **argv)
{
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipMalloc(&mem_d, 10000));
    
    std::thread th(threadfunc);
    th.join();
    std::cout << "Done\n";
}
